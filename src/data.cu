/**
 \file  data.cu
 \brief Implementation of functions to create and destroy CudaData struct
*/

#include <hiprand.h>
#include "data.cuh"
#include "discretization.cuh"

void init_cuda(struct HostData* host,
               const int nx, const int ny, const int nm,
               const fp_t* kappa, const fp_t* omega, const fp_t* Lmob,
               struct CudaData* dev)
{
	/* allocate memory on device */
	checkCuda(hipMalloc((void**) &(dev->prng), nx * ny * sizeof(hiprandState)));

	hipStreamCreate(&(dev->str_A));
	hipStreamCreate(&(dev->str_B));
	hipStreamCreate(&(dev->str_C));
	hipStreamCreate(&(dev->str_D));

	hipEventCreate(&(dev->ev_A));
	hipEventCreate(&(dev->ev_B));
	hipEventCreate(&(dev->ev_C));
	hipEventCreate(&(dev->ev_D));

	checkCuda(hipMalloc((void**) &(dev->conc_Cr_old), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->conc_Cr_new), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->conc_Nb_old), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->conc_Nb_new), nx * ny * sizeof(fp_t)));

	checkCuda(hipMalloc((void**) &(dev->phi_del_old), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->phi_del_new), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->phi_lav_old), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->phi_lav_new), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->phi),         nx * ny * sizeof(fp_t)));

	checkCuda(hipMalloc((void**) &(dev->conc_Cr_gam), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->conc_Cr_del), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->conc_Cr_lav), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->conc_Nb_gam), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->conc_Nb_del), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->conc_Nb_lav), nx * ny * sizeof(fp_t)));

	checkCuda(hipMalloc((void**) &(dev->conc_Ni), nx * ny * sizeof(fp_t)));

	checkCuda(hipMalloc((void**) &(dev->mob_gam_CrCr), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->mob_gam_CrNb), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->mob_gam_NbCr), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->mob_gam_NbNb), nx * ny * sizeof(fp_t)));

	checkCuda(hipMalloc((void**) &(dev->mob_del_CrCr), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->mob_del_CrNb), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->mob_del_NbCr), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->mob_del_NbNb), nx * ny * sizeof(fp_t)));

	checkCuda(hipMalloc((void**) &(dev->mob_lav_CrCr), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->mob_lav_CrNb), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->mob_lav_NbCr), nx * ny * sizeof(fp_t)));
	checkCuda(hipMalloc((void**) &(dev->mob_lav_NbNb), nx * ny * sizeof(fp_t)));

	/* transfer mask to protected memory on GPU */
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_mask), host->mask_lap[0], nm * nm * sizeof(fp_t)));

	/* transfer mobility data to protected memory on GPU */
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_Kapp), kappa, NP * sizeof(fp_t)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_Omeg), omega, NP * sizeof(fp_t)));
	checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_Lmob), Lmob,  NP * sizeof(fp_t)));

	/* transfer data from host in to GPU */
	checkCuda(hipMemcpy(dev->conc_Cr_old, host->conc_Cr_old[0], nx * ny * sizeof(fp_t),
	                     hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(dev->conc_Nb_old, host->conc_Nb_old[0], nx * ny * sizeof(fp_t),
	                     hipMemcpyHostToDevice));

	checkCuda(hipMemcpy(dev->phi_del_old, host->phi_del_old[0], nx * ny * sizeof(fp_t),
	                     hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(dev->phi_lav_old, host->phi_lav_old[0], nx * ny * sizeof(fp_t),
	                     hipMemcpyHostToDevice));
}

void free_cuda(struct CudaData* dev)
{
	/* free memory on device */
	checkCuda(hipFree(dev->prng));

	hipStreamDestroy(dev->str_A);
	hipStreamDestroy(dev->str_B);
	hipStreamDestroy(dev->str_C);
	hipStreamDestroy(dev->str_D);

	hipEventDestroy(dev->ev_A);
	hipEventDestroy(dev->ev_B);
	hipEventDestroy(dev->ev_C);
	hipEventDestroy(dev->ev_D);

	checkCuda(hipFree(dev->conc_Cr_old));
	checkCuda(hipFree(dev->conc_Cr_new));
	checkCuda(hipFree(dev->conc_Nb_old));
	checkCuda(hipFree(dev->conc_Nb_new));

	checkCuda(hipFree(dev->phi_del_old));
	checkCuda(hipFree(dev->phi_del_new));
	checkCuda(hipFree(dev->phi_lav_old));
	checkCuda(hipFree(dev->phi_lav_new));
	checkCuda(hipFree(dev->phi));

	checkCuda(hipFree(dev->conc_Cr_gam));
	checkCuda(hipFree(dev->conc_Cr_del));
	checkCuda(hipFree(dev->conc_Cr_lav));
	checkCuda(hipFree(dev->conc_Nb_gam));
	checkCuda(hipFree(dev->conc_Nb_del));
	checkCuda(hipFree(dev->conc_Nb_lav));

	checkCuda(hipFree(dev->conc_Ni));

	checkCuda(hipFree(dev->mob_gam_CrCr));
	checkCuda(hipFree(dev->mob_gam_CrNb));
	checkCuda(hipFree(dev->mob_gam_NbCr));
	checkCuda(hipFree(dev->mob_gam_NbNb));

	checkCuda(hipFree(dev->mob_del_CrCr));
	checkCuda(hipFree(dev->mob_del_CrNb));
	checkCuda(hipFree(dev->mob_del_NbCr));
	checkCuda(hipFree(dev->mob_del_NbNb));

	checkCuda(hipFree(dev->mob_lav_CrCr));
	checkCuda(hipFree(dev->mob_lav_CrNb));
	checkCuda(hipFree(dev->mob_lav_NbCr));
	checkCuda(hipFree(dev->mob_lav_NbNb));
}

void read_out_result(struct CudaData* dev, struct HostData* host,
                     const int nx, const int ny)
{
	hipEventSynchronize(dev->ev_A);
	hipEventSynchronize(dev->ev_B);
	hipEventSynchronize(dev->ev_C);
	hipEventSynchronize(dev->ev_D);

	checkCuda(hipMemcpy(host->conc_Cr_new[0], dev->conc_Cr_old, nx * ny * sizeof(fp_t),
	                     hipMemcpyDeviceToHost));
	checkCuda(hipMemcpy(host->conc_Nb_new[0], dev->conc_Nb_old, nx * ny * sizeof(fp_t),
	                     hipMemcpyDeviceToHost));
	checkCuda(hipMemcpy(host->phi_del_new[0], dev->phi_del_old, nx * ny * sizeof(fp_t),
	                     hipMemcpyDeviceToHost));
	checkCuda(hipMemcpy(host->phi_lav_new[0], dev->phi_lav_old, nx * ny * sizeof(fp_t),
	                     hipMemcpyDeviceToHost));
}
